/* 
 * hisqForce.cu 
 *
 * D. Bollweg
 *
 */

#include "hisqForce.h"
#include "staggeredPhases.h"

/// Method for printing results, only needed for testing.
template<class floatT, size_t HaloDepth, CompressionType comp>
void printResult(Gaugefield<floatT, true, HaloDepth, comp> &g_dev) {
    typedef GIndexer<All, HaloDepth> GInd;
    Gaugefield<floatT, false, HaloDepth, comp> g_host(g_dev.getComm());
    g_host = g_dev;
    gSite site = GInd::getSite(0,0,0,0);
    GSU3<floatT> mat = g_host.getAccessor().getLink(GInd::getSiteMu(site,3));
    rootLogger.info(mat.getLink00() ,  mat.getLink01() ,  mat.getLink02());
    rootLogger.info(mat.getLink10() ,  mat.getLink11() ,  mat.getLink12());
    rootLogger.info(mat.getLink20() ,  mat.getLink21() ,  mat.getLink22());
    rootLogger.info(" ");
    site = GInd::getSite(0,0,0,1);
    mat = g_host.getAccessor().getLink(GInd::getSiteMu(site,3));
    rootLogger.info(mat.getLink00() ,  mat.getLink01() ,  mat.getLink02());
    rootLogger.info(mat.getLink10() ,  mat.getLink11() ,  mat.getLink12());
    rootLogger.info(mat.getLink20() ,  mat.getLink21() ,  mat.getLink22());
    rootLogger.info(" ");
    site = GInd::getSite(0,0,0,2);
    mat = g_host.getAccessor().getLink(GInd::getSiteMu(site,3));
    rootLogger.info(mat.getLink00() ,  mat.getLink01() ,  mat.getLink02());
    rootLogger.info(mat.getLink10() ,  mat.getLink11() ,  mat.getLink12());
    rootLogger.info(mat.getLink20() ,  mat.getLink21() ,  mat.getLink22());
    rootLogger.info(" ");
    site = GInd::getSite(0,0,0,3);
    mat = g_host.getAccessor().getLink(GInd::getSiteMu(site,3));
    rootLogger.info(mat.getLink00() ,  mat.getLink01() ,  mat.getLink02());
    rootLogger.info(mat.getLink10() ,  mat.getLink11() ,  mat.getLink12());
    rootLogger.info(mat.getLink20() ,  mat.getLink21() ,  mat.getLink22());
    rootLogger.info(" ");
    site = GInd::getSite(0,0,1,1);
    mat = g_host.getAccessor().getLink(GInd::getSiteMu(site,1));
    rootLogger.info(mat.getLink00() ,  mat.getLink01() ,  mat.getLink02());
    rootLogger.info(mat.getLink10() ,  mat.getLink11() ,  mat.getLink12());
    rootLogger.info(mat.getLink20() ,  mat.getLink21() ,  mat.getLink22());
    rootLogger.info(" ");
  
    return;
}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp, bool lvl1>
contribution_3link<floatT, onDevice, HaloDepth, comp, lvl1>::contribution_3link(Gaugefield<floatT, onDevice,HaloDepth,comp> &GaugeIn, Gaugefield<floatT, onDevice, HaloDepth> &ForceIn) : _gaugeAccessor(GaugeIn.getAccessor()), _forceAccessor(ForceIn.getAccessor()) {}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp, bool lvl1>
__host__ __device__ GSU3<floatT> contribution_3link<floatT, onDevice, HaloDepth, comp, lvl1>::operator() (gSiteMu siteMu) {
    typedef GIndexer<All, HaloDepth> GInd;
    gSite site = GInd::getSite(siteMu.isite);
    return threeLinkContribution<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _smParams);
}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
contribution_lepagelink<floatT, onDevice, HaloDepth, comp>::contribution_lepagelink(Gaugefield<floatT, onDevice, HaloDepth, comp> &GaugeIn, Gaugefield<floatT, onDevice, HaloDepth> &ForceIn) : _gaugeAccessor(GaugeIn.getAccessor()), _forceAccessor(ForceIn.getAccessor()) {}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
__host__ __device__ GSU3<floatT> contribution_lepagelink<floatT, onDevice, HaloDepth, comp>::operator() (gSiteMu siteMu) {
    typedef GIndexer<All, HaloDepth> GInd;
    gSite site = GInd::getSite(siteMu.isite);
    return lepagelinkContribution<floatT, HaloDepth, comp> (_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c_lp);
}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp, int Part>
contribution_7link<floatT, onDevice, HaloDepth, comp, Part>::contribution_7link(Gaugefield<floatT, onDevice, HaloDepth, comp> &GaugeIn, Gaugefield<floatT, onDevice, HaloDepth> &ForceIn) : _gaugeAccessor(GaugeIn.getAccessor()), _forceAccessor(ForceIn.getAccessor()) {}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp, int Part>
__host__ __device__ GSU3<floatT> contribution_7link<floatT, onDevice, HaloDepth, comp, Part>::operator() (gSiteMu siteMu) {
    typedef GIndexer<All, HaloDepth> GInd;
    gSite site = GInd::getSite(siteMu.isite);
    switch (Part) {
    case 1:
        return sevenLinkContribution_1<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c7);
    case 2:
        return sevenLinkContribution_2<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c7);
    case 3:
        return sevenLinkContribution_3<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c7);
    case 4:
        return sevenLinkContribution_4<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c7);
    case 5:
        return sevenLinkContribution_5<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c7);
    case 6:
        return sevenLinkContribution_6<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c7);
    case 7:
        return sevenLinkContribution_7<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c7);
    default:
        return gsu3_zero<floatT>();
    }
    
}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp, int part>
contribution_5link<floatT, onDevice, HaloDepth, comp, part>::contribution_5link(Gaugefield<floatT, onDevice, HaloDepth, comp> &GaugeIn, Gaugefield<floatT, onDevice, HaloDepth> &ForceIn) : _gaugeAccessor(GaugeIn.getAccessor()), _forceAccessor(ForceIn.getAccessor()) {}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp, int part>
__host__ __device__ GSU3<floatT> contribution_5link<floatT, onDevice, HaloDepth, comp, part>::operator() (gSiteMu siteMu) {
    typedef GIndexer<All, HaloDepth> GInd;
    gSite site = GInd::getSite(siteMu.isite);
    switch (part) {
    case 11: return fiveLinkContribution_11<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c5);
    case 12: return fiveLinkContribution_12<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c5);
    case 13: return fiveLinkContribution_13<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c5);
    case 14: return fiveLinkContribution_14<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c5);
    case 20: return fiveLinkContribution_20<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c5);
    case 30: return fiveLinkContribution_30<floatT, HaloDepth, comp>(_gaugeAccessor, _forceAccessor, site, siteMu.mu, _c5);
    default: return gsu3_zero<floatT>();
    }
}


template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
constructU3ProjForce<floatT, onDevice, HaloDepth, comp>::constructU3ProjForce(Gaugefield<floatT, onDevice, HaloDepth> &GaugeIn, Gaugefield<floatT, onDevice, HaloDepth, comp> &ForceIn) : _gaugeAccessor(GaugeIn.getAccessor()), _forceAccessor(ForceIn.getAccessor()) {}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
__host__ __device__ GSU3<floatT> constructU3ProjForce<floatT, onDevice, HaloDepth, comp>::operator()(gSiteMu siteMu) {
    typedef GIndexer<All, HaloDepth> GInd;
    gSite site = GInd::getSite(siteMu.isite);
    return derivativeProjectU3<floatT, HaloDepth>(_gaugeAccessor, _forceAccessor, site, siteMu.mu);
}



template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
finalizeForce<floatT, onDevice, HaloDepth, comp>::finalizeForce(Gaugefield<floatT, onDevice, HaloDepth,comp> &GaugeIn, Gaugefield<floatT, onDevice, HaloDepth, comp> &ForceIn) : _gaugeAccessor(GaugeIn.getAccessor()), _forceAccessor(ForceIn.getAccessor()) {}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
__host__ __device__ GSU3<floatT> finalizeForce<floatT, onDevice, HaloDepth, comp>::operator()(gSiteMu siteMu) {
    typedef GIndexer<All, HaloDepth> GInd;
    GSU3<floatT> tmp = floatT(2.0)*_gaugeAccessor.getLink(siteMu)*_forceAccessor.getLink(siteMu);
    tmp.TA();
  
    return tmp;
}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
constructNaikDerivForce<floatT, onDevice, HaloDepth, comp>::constructNaikDerivForce(Gaugefield<floatT, onDevice, HaloDepth> &GaugeIn, Gaugefield<floatT, onDevice, HaloDepth, comp> &ForceIn) : _gaugeAccessor(GaugeIn.getAccessor()), _forceAccessor(ForceIn.getAccessor()) {}

template<class floatT, bool onDevice, size_t HaloDepth, CompressionType comp>
__host__ __device__ GSU3<floatT> constructNaikDerivForce<floatT, onDevice, HaloDepth, comp>::operator()(gSiteMu siteMu) {
    typedef GIndexer<All, HaloDepth> GInd;
    gSite site = GInd::getSite(siteMu.isite);
    return naikLinkDerivative<floatT, HaloDepth>(_gaugeAccessor,_forceAccessor,site, siteMu.mu);
}

template<class floatT,bool onDevice, Layout LatticeLayout, size_t HaloDepthSpin, size_t NStacks>
struct multiplySimpleArraySpinor {

    SimpleArray<floatT, NStacks> _Arr;
    gVect3arrayAcc<floatT> _Spinor;
    multiplySimpleArraySpinor(SimpleArray<floatT, NStacks>& Arr, Spinorfield<floatT,onDevice,LatticeLayout,HaloDepthSpin,NStacks>& Spinor) : _Arr(Arr), _Spinor(Spinor.getAccessor()) {}
  
    __host__ __device__ void initialize(__attribute__((unused)) gSite& site) {
    }
  
    __host__ __device__ gVect3<floatT> operator()(gSite& site, size_t stack) {
  
        gSiteStack siteStack = GIndexer<LatticeLayout,HaloDepthSpin>::getSiteStack(site,stack);
        gVect3<floatT> tmp;
        tmp = _Spinor.getElement(siteStack);
        tmp = tmp * _Arr[stack];
        return tmp;
    }
};

template<class floatT, bool onDevice, size_t HaloDepth, size_t HaloDepthSpin, int steps, bool runTesting, const int rdeg>
tensor_product<floatT, onDevice, HaloDepth, HaloDepthSpin, steps, runTesting,rdeg>::tensor_product(Gaugefield<floatT, onDevice,  HaloDepth> &gaugeIn,
                                                   gVect3arrayAcc<floatT> x, gVect3arrayAcc<floatT> y,
                                                   SimpleArray<floatT, rdeg> rat_num)
  : _x(x), _y(y), gAccessor(gaugeIn.getAccessor()), _rat_num(rat_num) {}

template<class floatT, bool onDevice, size_t HaloDepth, size_t HaloDepthSpin, int steps, bool runTesting, const int rdeg>
__host__ __device__ GSU3<floatT> tensor_product<floatT, onDevice, HaloDepth, HaloDepthSpin, steps, runTesting,rdeg>::operator()(gSiteMu site) {
    typedef GIndexer<All, HaloDepthSpin> GInd_All;
    typedef GIndexer<Even,HaloDepthSpin> GInd_even;
    typedef GIndexer<Odd, HaloDepthSpin> GInd_odd;
    
    
    floatT actConstant;
    if (steps == 1) {
        actConstant = 1./2.; //c1000
    } else if (steps == 3) {
        actConstant = -1./48.; //c3000
    }
    
    
    GSU3<floatT> tmp = gsu3_zero<floatT>();
    for (int i = 0; i < rdeg; i++) {
        sitexyzt here =  site.coord;
    
        bool oddness = (isOdd(here.x) ^ isOdd(here.y)) ^ (isOdd(here.z) ^ isOdd(here.t));
    
        if (!oddness) {
            gSiteStack even_site = GInd_even::getSiteStack(here.x, here.y, here.z, here.t,i);
            gSiteStack odd_site = GInd_even::template site_move<steps>(even_site,site.mu);
      
            tmp += _rat_num[i]*tensor_prod(_y.getElement(odd_site),conj(_x.getElement(even_site)));

        } else {
            gSiteStack odd_site = GInd_odd::getSiteStack(here.x, here.y, here.z, here.t,i);
            gSiteStack even_site = GInd_odd::template site_move<steps>(odd_site,site.mu);
      
            tmp -= _rat_num[i]*tensor_prod(_x.getElement(even_site), conj(_y.getElement(odd_site)));
        }
    }
  
    return actConstant*tmp;
}

template<class floatT, bool onDevice, size_t HaloDepth, size_t HaloDepthSpin,  CompressionType comp, bool runTesting, const int rdeg>
HisqForce<floatT, onDevice, HaloDepth, HaloDepthSpin, comp, runTesting, rdeg>::
HisqForce(Gaugefield<floatT, onDevice,  HaloDepth,R18> &GaugeBase,
      Gaugefield<floatT, onDevice,  HaloDepth, comp> &Force,
      AdvancedMultiShiftCG<floatT, rdeg> &cg,
      HisqDSlash<floatT, onDevice, Even, HaloDepth, HaloDepthSpin, 1> &dslash,
      HisqDSlash<floatT, onDevice, Even, HaloDepth, HaloDepthSpin, rdeg> &dslash_multi,
      RhmcParameters &rhmc_param,
      RationalCoeff &rat,
      HisqSmearing<floatT, onDevice, HaloDepth, R18, R18, R18, U3R14> &smearing)
: _GaugeU3P(GaugeBase.getComm(),"SHARED_GAUGELVL2"),
  _GaugeLvl1(GaugeBase.getComm(),"SHARED_GAUGENAIK"),
  _TmpForce(GaugeBase.getComm()),
  _GaugeBase(GaugeBase),
  _spinor_x(GaugeBase.getComm()),
  _spinor_y(GaugeBase.getComm(), "SHARED_tmp"),
  _Dummy(GaugeBase.getComm(), "SHARED_DUMMY"),
  _createF2(_GaugeLvl1,_TmpForce),
  _createNaikF1(_GaugeU3P,_TmpForce), _finalizeF3(_GaugeU3P,_TmpForce),
  _cg(cg), _dslash(dslash), _dslash_multi(dslash_multi), _smearing(smearing),
  _rhmc_param(rhmc_param), _rat(rat),
  F1_create_3Link(_GaugeU3P,Force),F1_7link_part_1(_GaugeU3P,Force), F1_7link_part_2(_GaugeU3P,Force), F1_7link_part_3(_GaugeU3P,Force), F1_7link_part_4(_GaugeU3P,Force) , F1_7link_part_5(_GaugeU3P,Force), F1_7link_part_6(_GaugeU3P,Force), F1_7link_part_7(_GaugeU3P,Force), F1_5link_part11(_GaugeU3P,Force), F1_5link_part12(_GaugeU3P,Force), F1_5link_part13(_GaugeU3P,Force), F1_5link_part14(_GaugeU3P,Force),F1_5link_part20(_GaugeU3P,Force),  F1_5link_part30(_GaugeU3P,Force), F1_lepagelink(_GaugeU3P, Force),
  F3_create_3Link(_GaugeU3P,Force), F3_7link_part_1(_GaugeU3P,Force), F3_7link_part_2(_GaugeU3P,Force), F3_7link_part_3(_GaugeU3P,Force), F3_7link_part_4(_GaugeU3P,Force) , F3_7link_part_5(_GaugeU3P,Force), F3_7link_part_6(_GaugeU3P,Force), F3_7link_part_7(_GaugeU3P,Force), F3_5link_part11(_GaugeU3P,Force), F3_5link_part12(_GaugeU3P,Force), F3_5link_part13(_GaugeU3P,Force), F3_5link_part14(_GaugeU3P,Force),F3_5link_part20(_GaugeU3P,Force),  F3_5link_part30(_GaugeU3P,Force)  {}

template<class floatT, bool onDevice, size_t HaloDepth, size_t HaloDepthSpin, CompressionType comp, bool runTesting, const int rdeg>
void HisqForce<floatT,onDevice, HaloDepth, HaloDepthSpin, comp, runTesting,rdeg>::make_f0(Spinorfield<floatT, onDevice, Even, HaloDepthSpin> &SpinorIn,
                              Gaugefield<floatT, onDevice,  HaloDepth, comp> &Force,
                              Gaugefield<floatT, onDevice,  HaloDepth, comp> &NaikForce,
                              bool isLight)
{

    Force.iterateWithConst(gsu3_zero<floatT>());
    NaikForce.iterateWithConst(gsu3_zero<floatT>());
    SimpleArray<floatT, rdeg> shifts;
    SimpleArray<floatT, rdeg> rat_num;
  
    if (isLight) {
        shifts[0] =_rat.r_bar_lf_den[0] + _rhmc_param.m_ud()*_rhmc_param.m_ud();
        rat_num[0] = _rat.r_bar_lf_num[0];
        for (int i = 1; i < rdeg; i++) {
            shifts[i] = _rat.r_bar_lf_den[i] -_rat.r_bar_lf_den[0];
            rat_num[i] = _rat.r_bar_lf_num[i];
        }
    } else {
        shifts[0] =_rat.r_bar_sf_den[0] + _rhmc_param.m_s()*_rhmc_param.m_s();
        rat_num[0] = _rat.r_bar_sf_num[0];
        for (int i = 1; i < rdeg; i++) {
            shifts[i] = _rat.r_bar_sf_den[i] -_rat.r_bar_sf_den[0];
            rat_num[i] = _rat.r_bar_sf_num[i];
        }
    }
  
  
    _cg.invert(_dslash,_spinor_x,SpinorIn,shifts,_rhmc_param.cgMax(),_rhmc_param.residue_force());
    
    _dslash_multi.Dslash(_spinor_y,_spinor_x,true);
  
    Force.iterateOverBulkAllMu(tensor_product<floatT, onDevice, HaloDepth, HaloDepthSpin, 1, runTesting>(Force, _spinor_x.getAccessor(), _spinor_y.getAccessor(),rat_num));
    
    Force.updateAll();
    _TmpForce.iterateOverBulkAllMu(tensor_product<floatT, onDevice, HaloDepth, HaloDepthSpin, 3, runTesting>(NaikForce, _spinor_x.getAccessor(), _spinor_y.getAccessor(),rat_num));
  
    
    _TmpForce.updateAll();
    return;
}

template<class floatT, bool onDevice, size_t HaloDepth, size_t HaloDepthSpin, CompressionType comp, bool runTesting, const int rdeg>
void HisqForce<floatT,onDevice, HaloDepth,HaloDepthSpin,comp,runTesting,rdeg>::TestForce(Spinorfield<floatT, onDevice, Even, HaloDepthSpin> &SpinorIn, Gaugefield<floatT,onDevice,HaloDepth,comp> &Force, grnd_state<true> &d_rand) {
    
    if (!runTesting) {
        rootLogger.error("Calling member function TestForce should only be used when the template parameter runTesting is set to true!");
    }
    
    Force.iterateWithConst(gsu3_zero<floatT>());
    _TmpForce.iterateWithConst(gsu3_zero<floatT>());
  
    Spinorfield<floatT, onDevice, Even, HaloDepthSpin> temp(SpinorIn.getComm());
    Spinorfield<floatT, false, Even, HaloDepthSpin> SpinorOutput(SpinorIn.getComm());
    
    gVect3<floatT> SpinorTestOutput;
    for (int i = 0; i < rdeg; i++) {
        temp.gauss(d_rand.state);
        _spinor_x.copyFromStackToStack(temp,i,0);
        SpinorOutput = temp;
        SpinorTestOutput = SpinorOutput.getAccessor().getElement(GIndexer<Even,HaloDepthSpin>::getSite(0,0,0,0));
        rootLogger.info("Xi_" ,  i ,  " " ,  SpinorTestOutput);
    }
  
    SimpleArray<floatT, rdeg> rat_num;
  
    for (int i = 0; i < rdeg; i++) {
        rat_num[i] = _rat.r_inv_lf_num[i];
    }
  
    _dslash_multi.Dslash(_spinor_y,_spinor_x);
    Force.iterateOverBulkAllMu(tensor_product<floatT, onDevice, HaloDepth, HaloDepthSpin, 1, runTesting>(Force, _spinor_x.getAccessor(), _spinor_y.getAccessor(),rat_num));
    _TmpForce.iterateOverBulkAllMu(tensor_product<floatT, onDevice, HaloDepth, HaloDepthSpin, 3, runTesting>(_TmpForce, _spinor_x.getAccessor(), _spinor_y.getAccessor(),rat_num));
    
    Force.updateAll();
    _TmpForce.updateAll();
   
    rootLogger.info("f0 intermediate result");
    printResult<floatT,HaloDepth,comp>(_TmpForce);
    _smearing.template SmearLvl1<R18>(_GaugeLvl1);
    _smearing.template ProjectU3<R18,R18>(_GaugeLvl1,_GaugeU3P);
  
    printResult<floatT,HaloDepth,R18>(_GaugeU3P);
  
    staggeredPhaseKernel<floatT, onDevice, HaloDepth,R18> multPhase(_GaugeU3P,_rhmc_param.mu_f());
    _GaugeU3P.iterateOverBulkAllMu(multPhase);
    
    _Dummy.template iterateOverBulkAllMu<64>(_createNaikF1);
    _TmpForce = _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_create_3Link);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part11);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part12);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part13);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part14);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part20);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part30);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_lepagelink);
    _TmpForce = _TmpForce + _Dummy;
    
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_1);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_2);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_3);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_4);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_5);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_6);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_7);
    _TmpForce = _TmpForce + _Dummy;
    
   
    rootLogger.info("f1 intermediate result");
    printResult<floatT,HaloDepth,comp>(_TmpForce);
    _TmpForce.updateAll();
  
    staggeredPhaseKernel<floatT,onDevice, HaloDepth,R18> multPhaselv1(_GaugeLvl1,_rhmc_param.mu_f());
    _GaugeLvl1.iterateOverBulkAllMu(multPhaselv1); 
  
    Force.iterateOverBulkAllMu(_createF2);
    Force.updateAll();
    
    rootLogger.info("f2 intermediate result");
    printResult<floatT,HaloDepth,comp>(Force);
    staggeredPhaseKernel<floatT,onDevice, HaloDepth,R18> multPhaseB(_GaugeBase,_rhmc_param.mu_f());
    _GaugeU3P.iterateOverBulkAllMu(multPhaseB); //reuse U3P Field here
    
   
    _TmpForce.template iterateOverBulkAllMu<64>(F3_create_3Link);
    _Dummy.iterateOverBulkAllMu(F3_5link_part11);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part12);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part13);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part14);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part20);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part30);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_1);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_2);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_3);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_4);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_5);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_6);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_7);
    _TmpForce = _TmpForce + _Dummy;
    rootLogger.info("f3 intermediate result");
    printResult<floatT,HaloDepth,comp>(_TmpForce);
  
    Force.iterateOverBulkAllMu(_finalizeF3);
    
    rootLogger.info("f3 final result");
    printResult<floatT,HaloDepth,comp>(Force);
    
    return;
}


template<class floatT, bool onDevice, size_t HaloDepth, size_t HaloDepthSpin, CompressionType comp, bool runTesting, const int rdeg>
void HisqForce<floatT,onDevice, HaloDepth, HaloDepthSpin, comp, runTesting, rdeg>::updateForce(Spinorfield<floatT,onDevice,Even,HaloDepthSpin> &SpinorIn,
                                  Gaugefield<floatT,onDevice,HaloDepth,comp> &Force,
                                  bool isLight) {
    make_f0(SpinorIn,Force,_TmpForce,isLight);
    _smearing.template SmearLvl1<R18>(_GaugeLvl1);
    
    _smearing.template ProjectU3<R18,R18>(_GaugeLvl1,_GaugeU3P);
    
    staggeredPhaseKernel<floatT, onDevice, HaloDepth,R18> multPhase(_GaugeU3P,_rhmc_param.mu_f());
    _GaugeU3P.iterateOverBulkAllMu(multPhase);
   
    _GaugeU3P.updateAll();
    
    _Dummy.iterateOverBulkAllMu(_createNaikF1);
    _TmpForce = _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_create_3Link);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part11);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part12);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part13);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part14);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part20);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_5link_part30);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F1_lepagelink);
    _TmpForce = _TmpForce + _Dummy;
    
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_1);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_2);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_3);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_4);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_5);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_6);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F1_7link_part_7);
    _TmpForce = _TmpForce + _Dummy;
  
    
    
    staggeredPhaseKernel<floatT,onDevice, HaloDepth,R18> multPhaselv1(_GaugeLvl1, _rhmc_param.mu_f());
    _GaugeLvl1.iterateOverBulkAllMu(multPhaselv1); 
    
    Force.iterateOverBulkAllMu(_createF2);
    Force.updateAll();
    
    staggeredPhaseKernel<floatT,onDevice, HaloDepth,R18> multPhaseB(_GaugeBase,_rhmc_param.mu_f());
    _GaugeU3P.iterateOverBulkAllMu(multPhaseB); // reuse U3P Field here
    
    _GaugeU3P.updateAll();
    
    _TmpForce.template iterateOverBulkAllMu<64>(F3_create_3Link);
    _Dummy.iterateOverBulkAllMu(F3_5link_part11);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part12);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part13);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part14);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part20);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.iterateOverBulkAllMu(F3_5link_part30);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_1);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_2);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_3);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_4);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_5);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_6);
    _TmpForce = _TmpForce + _Dummy;
    _Dummy.template iterateOverBulkAllMu<64>(F3_7link_part_7);
    _TmpForce = _TmpForce + _Dummy;
    
    Force.iterateOverBulkAllMu(_finalizeF3);
    
    return;
}

#define HFORCE_INIT(floatT, HALO, HALOSPIN)			       \
  template class HisqForce<floatT, true, HALO, HALOSPIN, R18, false>; \
  template class HisqForce<floatT, true, HALO, HALOSPIN, R18, true>;
INIT_PHHS(HFORCE_INIT)

